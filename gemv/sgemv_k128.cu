#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <math.h>

#define FLOAT4(value) *(float4*)(&(value))
#define OFFSET(row, col, ld) ((row) * (ld) + (col))

#define checkCudaErrors(func)				\
{									\
    hipError_t e = (func);			\
    if(e != hipSuccess)						                \
        printf ("%s %d CUDA: %s\n", __FILE__,  __LINE__, hipGetErrorString(e));		\
}

template <unsigned int WarpSize>
__device__ __forceinline__ float warpReduceSum(float sum)
{
    if (warpSize >= 32) sum += __shfl_down_sync(0xffffffff, sum, 16); // 0-16, 1-17, 2-18, etc.
    if (warpSize >= 16) sum += __shfl_down_sync(0xffffffff, sum, 8 ); // 0-8, 1-9, 2-10, etc.
    if (warpSize >= 8)  sum += __shfl_down_sync(0xffffffff, sum, 4 ); // 0-4, 1-5, 2-6, etc.
    if (warpSize >= 4)  sum += __shfl_down_sync(0xffffffff, sum, 2 ); // 0-2, 1-3, 4-6, 5-7, etc.
    if (warpSize >= 2)  sum += __shfl_down_sync(0xffffffff, sum, 1 ); // 0-1, 2-3, 4-5, etc.

    return sum;
}

__global__ void Sgemv_k128(float* A, float* x, float* y, const int M, const int K)
{
    const int warp_size = 32;
    int laneId = threadIdx.x % warp_size;
    int current_row = blockDim.y * blockIdx.x + threadIdx.y;

    if (current_row >= M) return;

    float res = 0.0f;
    int kIteration = (K / warp_size) / 4;
    if (kIteration == 0) kIteration = 1;

    #pragma unroll
    for (int i = 0; i < kIteration; i++)
    {
        int current_col = (i * warp_size + laneId);
        float4 tmp_A = reinterpret_cast<float4 *>(&A[current_row * K])[current_col];
        float4 tmp_x = reinterpret_cast<float4 *>(x)[current_col];
        res += tmp_A.x * tmp_x.x;
        res += tmp_A.y * tmp_x.y;
        res += tmp_A.z * tmp_x.z;
        res += tmp_A.w * tmp_x.w;
    }

    res = warpReduceSum<warp_size>(res);

    if (laneId == 0) y[current_row] = res;
}

int main(int argc, char** argv)
{
    size_t M = 1024;
    // size_t M = 16384;
    size_t K = 128;

    size_t bytes_A = sizeof(float) * M * K;
    size_t bytes_x = sizeof(float) * K;
    size_t bytes_y = sizeof(float) * M;

    float* h_A  = (float*)malloc(bytes_A);
    float* h_x  = (float*)malloc(bytes_x);
    float* h_y  = (float*)malloc(bytes_y);
    float* h_y1 = (float*)malloc(bytes_y);

    float* d_A;
    float* d_x;
    float* d_y;

    checkCudaErrors(hipMalloc(&d_A, bytes_A));
    checkCudaErrors(hipMalloc(&d_x, bytes_x));
    checkCudaErrors(hipMalloc(&d_y, bytes_y));

    double duration[2] = {0.0f, 0.0f};
    double GFLOPS[2]   = {0.0f, 0.0f};
    double GFLOPs      = 2.0 * M * 1 * K;

    // 生成A的数据
    for (int i = 0; i < M * K; i++)
    {
        h_A[i] = (float)i / K;
    }

    // 生成x的数据
    for (int i = 0; i < K; i++)
    {
        h_x[i] = 1;
    }
    
    memset(h_y, 0, M * sizeof(float));
    memset(h_y1, 0, M * sizeof(float));

    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    float mseTotal = 0.0f;
    int nIter = 1000;

    checkCudaErrors(hipMemcpy(d_A, h_A, bytes_A, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_x, h_x, bytes_x, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_y, h_y, bytes_y, hipMemcpyHostToDevice));

    checkCudaErrors(hipEventRecord(start));

    for (int run = 0; run < nIter; run++)
    {
        dim3 dimGrid(M / 4);
        dim3 dimBlock(32, 4);
        Sgemv_k128<<<dimGrid, dimBlock>>>(d_A, d_x, d_y, M, K);
    }

    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&mseTotal, start, stop));
    checkCudaErrors(hipMemcpy(h_y, d_y, bytes_y, hipMemcpyDeviceToHost));

    duration[0] = mseTotal / nIter;
    GFLOPS[0]   = (GFLOPs *1.0e-9f) / (duration[0] / 1000.0f);
    printf( "My gemm Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
        GFLOPS[0],
        duration[0],
        GFLOPs);
    
    // cublas
    hipblasHandle_t blas_handle;
    hipblasCreate(&blas_handle);
    float alpha = 1.0f;
    float beta  = 0.0f;
    checkCudaErrors(hipMemcpy(d_y, h_y1, bytes_y, hipMemcpyHostToDevice));

    checkCudaErrors(hipEventRecord(start));
    for (int run = 0; run < nIter; run++)
    {
        hipblasSgemv(
            blas_handle, HIPBLAS_OP_T,
            K, M, &alpha,
            d_A, K, d_x, 1, &beta, d_y, 1
        );
    }

    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&mseTotal, start, stop));

    checkCudaErrors(hipMemcpy(h_y1, d_y, bytes_y, hipMemcpyDeviceToHost));

    duration[1] = mseTotal / nIter;
    GFLOPS[1]   = (GFLOPs * 1.0e-9f) / (duration[1] / 1000.0f);
    printf( "CuBlas Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
        GFLOPS[1],
        duration[1],
        GFLOPs);
    
    hipblasDestroy(blas_handle);

    double eps = 1.e-6; // match zero
    bool correct = true;
    for (int i = 0; i < M; i++)
    {
        double abs_err = fabs(h_y[i] - h_y1[i]);
        double dot_length = M;
        double abs_val = fabs(h_y[i]);
        double rel_err = abs_err / abs_val / dot_length;
        if (rel_err > eps)
        {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n",
                    i, h_y[i], h_y1[i], eps);
            correct = false;
            break;
        }
    }
    
    printf("%s\n", correct ? "Result= PASS" : "Result= FAIL");
    printf("My Gemm to CuBlas implementation ratio = %f\n", GFLOPS[0] / GFLOPS[1]);

    // Free Memory
    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);

    free(h_A);
    free(h_x);
    free(h_y);
}