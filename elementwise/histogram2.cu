#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <assert.h>
#include <algorithm>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#define INT4(value)    *(int4*)(&(value))
#define FLOAT4(value)  *(float4*)(&(value))

#define checkCudaErrors(func)               \
{                                   \
    hipError_t e = (func);         \
    if(e != hipSuccess)                                        \
        printf ("%s %d CUDA: %s\n", __FILE__,  __LINE__, hipGetErrorString(e));        \
}

template <typename T>
inline T CeilDiv(const T& a, const T& b)
{
    return (a + b - 1) / b;
}

#define DATA_LEN (100 * 1024 * 1024)

inline int rnd(float x)
{
    return static_cast<int>(x * rand() / RAND_MAX);
}

__global__ void cal_hist(unsigned char* buffer, unsigned int* hist, long data_size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    while (index < data_size)
    {
        atomicAdd(&hist[buffer[index]],1);
        index += stride;
    }
}

__global__ void cal_hist2(unsigned char* buffer, unsigned int* hist, long data_size)
{
    __shared__ unsigned int temp[256];
    temp[threadIdx.x] = 0;
    __syncthreads();

    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    while (index < data_size)
    {
        atomicAdd(&temp[buffer[index]], 1);
        index += stride;
    }
    __syncthreads();

    atomicAdd(&(hist[threadIdx.x]), temp[threadIdx.x]);
}

int main(int argc, char** argv)
{
    unsigned char* buffer = new unsigned char[DATA_LEN];
    for (int i = 0; i < DATA_LEN; ++i)
    {
        buffer[i] = rnd(255);
        if (buffer[i] > 255)
        {
            printf("error\n");
        }
    }

    unsigned int* d_hist;
    checkCudaErrors(hipMalloc((void**)&d_hist, sizeof(unsigned int) * 256));
    checkCudaErrors(hipMemset(d_hist, 0, sizeof(int)));

    unsigned char* d_buffer;
    checkCudaErrors(hipMalloc((void**)&d_buffer, sizeof(unsigned char) * DATA_LEN));
    checkCudaErrors(hipMemcpy(d_buffer, buffer, sizeof(unsigned char) * DATA_LEN, hipMemcpyHostToDevice));

    hipDeviceProp_t prop;
    checkCudaErrors(hipGetDeviceProperties(&prop, 0));
    int block_num = prop.multiProcessorCount;

    hipEvent_t start, end;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&end));
    checkCudaErrors(hipEventRecord(start, 0));

    int block_size = 256;
    // cal_hist<<<block_num, 256>>>(d_buffer, d_hist, DATA_LEN); // 30.726688 ms
    // cal_hist2<<<block_num, 256>>>(d_buffer, d_hist, DATA_LEN);   // 4.517536  ms
    cal_hist2<<<CeilDiv(DATA_LEN, block_size), block_size>>>(d_buffer, d_hist, DATA_LEN);

    float elapsed_time;
    checkCudaErrors(hipEventRecord(end, 0));
    checkCudaErrors(hipEventSynchronize(end));
    checkCudaErrors(hipEventElapsedTime(&elapsed_time, start, end));
    printf("using time: %f ms\n", elapsed_time);

    unsigned int h_hist[256];
    checkCudaErrors(hipMemcpy(h_hist, d_hist, sizeof(unsigned int) * 256, hipMemcpyDeviceToHost));

    long hist_count = 0;
    for (int i = 0; i <256; ++i)
    {
        hist_count += h_hist[i];
    }
    printf("histogram sum: %d\n", hist_count);

    for (int i = 0; i < DATA_LEN; ++i)
    {
        h_hist[buffer[i]]--;
    }
    for (int i = 0; i < 256; ++i)
    {
        if (h_hist[i] != 0)
        {
            printf("cal error\n");
        }
    }
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(end));
    checkCudaErrors(hipFree(d_hist));
    checkCudaErrors(hipFree(d_buffer));

    delete[] buffer;

    return 0;

}