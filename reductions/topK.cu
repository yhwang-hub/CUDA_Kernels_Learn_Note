
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define N 100000000
#define BLOCK_SIZE 256
#define GRID_SIZE 64
#define topk 20

__managed__ int source[N];
__managed__ int gpu_result[topk];
__managed__ int _1_pass_result[topk * GRID_SIZE];

// topK == 20
// source[N]:  1 + 2 + 3 + 4 + ...............N   
// cpu: for loop 
// gpu: 1 + 2 + 3 + 4 + ...............N    0 + 1 + 2 + 3 + 4[20] + 5 + 6 + 7 
// thread id step 0:  tid0:source[0][20] > source[4][20]? source[0] & source[4]-> source[0][20]
//                    tid1:source[1] + source[5] -> source[1]
//                    tid2:source[2] + source[6] -> source[2]
//                    tid4:source[4] + source[7] -> source[3]
//           step 1:  tid0: source[0] + source[2] -> source[0]
//                    tid1: source[1] + source[3] -> source[1]
//
//           step 2:  tid0: source[0] + source[1] -> source[0]
// thread id: blockDim.x * blockIdx.x + threadIdx.x + step * blockDim.x * GridDim.x
// thread 0: source[0, 8, 16, 24] sum -> shared memory

__device__ __host__ void insert_value(int *array, int k, int data)
{
    for(int i=0; i<k; i++)
    {
        if(array[i] == data)
        {
            return;
        }
    }
    if(data < array[k-1])
    {
        return;
    }
    //19, 18, 17, 16,.........4, 3, 2, 1, 0
    for(int i = k-2; i>=0; i--)
    {
        if(data > array[i])
        {
            array[i + 1] = array[i];
        }
        else
        {
            array[i+1] = data;
            return;
        }
    }
    
    array[0] = data;
}

__global__ void gpu_topk(int *input, int *output, int length, int k)
{
    __shared__ int ken[BLOCK_SIZE * topk];
    int top_array[topk];

    for(int i = 0; i<topk; i++)
    {
        top_array[i] = INT_MIN;
    }

    for(int idx = threadIdx.x + blockDim.x * blockIdx.x; idx < length; idx += gridDim.x * blockDim.x)
    {
        insert_value(top_array, topk, input[idx]);
    }
    for(int i =0; i<topk; i++)
    {
        ken[topk * threadIdx.x + i] = top_array[i];
    }
    __syncthreads();

    for(int i = BLOCK_SIZE/2; i>=1; i/=2)
    {
        if(threadIdx.x < i)
        {
            for(int m=0; m<topk; m++)
            {
                insert_value(top_array, topk, ken[topk *(threadIdx.x + i) + m]);
            }
        }
        __syncthreads();
        if(threadIdx.x < i)
        {
            for(int m=0; m<topk; m++)
            {
                ken[topk * threadIdx.x + m] = top_array[m];
            }
        }
        __syncthreads();
    }
    if(blockIdx.x * blockDim.x < length)
    {
        if(threadIdx.x == 0 )
        {
            for(int i =0; i < topk; i++)
            {
                output[topk * blockIdx.x + i] = ken[i];
            }
        }
    }


}

void cpu_topk(int *input, int *output, int length, int k)
{
    for(int i =0; i< length; i++)
    {
        insert_value(output, k, input[i]);
    }
}

int main()
{
    printf("Init source data...........\n");
    for(int i=0; i<N; i++)
    {
        source[i] = rand();
    }

    printf("Complete init source data.....\n");
    hipEvent_t start, stop_gpu, stop_cpu;
    hipEventCreate(&start);
    hipEventCreate(&stop_gpu);
    hipEventCreate(&stop_cpu);

    hipEventRecord(start);
    hipEventSynchronize(start);
    printf("GPU Run **************\n");
    for(int i =0; i<20; i++)
    {
        gpu_topk<<<GRID_SIZE, BLOCK_SIZE>>>(source, _1_pass_result, N, topk);

        gpu_topk<<<1, BLOCK_SIZE>>>(_1_pass_result, gpu_result, topk * GRID_SIZE, topk);

        hipDeviceSynchronize();
    }
    printf("GPU Complete!!!\n");
    hipEventRecord(stop_gpu);
    hipEventSynchronize(stop_gpu);
    
    int cpu_result[topk] ={0};
    printf("CPU RUN***************\n");
    cpu_topk(source, cpu_result, N, topk);
    hipEventRecord(stop_cpu);
    hipEventSynchronize(stop_cpu);
    printf("CPU Complete!!!!!\n");

    float time_cpu, time_gpu;
    hipEventElapsedTime(&time_gpu, start, stop_gpu);
    hipEventElapsedTime(&time_cpu, stop_gpu, stop_cpu);

    bool error = false;
    for(int i =0; i<topk; i++)
    {
        printf("CPU top%d: %d; GPU top%d: %d;\n", i+1, cpu_result[i], i+1, gpu_result[i]);
        if(fabs(gpu_result[i] - cpu_result[i]) > 0)
        {
            error = true;
        }
    }
    printf("Result: %s\n", (error?"Error":"Pass"));
    printf("CPU time: %.2f ms; GPU time: %.2f ms\n", time_cpu, (time_gpu/20.0));
}